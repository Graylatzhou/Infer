#include "hip/hip_runtime.h"
#include "operators/embedding.hpp"

#define FLOAT4_OUTPUT(value) (reinterpret_cast<float4 *>(&(value))[0])
#define FLOAT4_INPUT(value) (reinterpret_cast<const float4 *>(&(value))[0])
#define BFLOAT4_OUTPUT(value) (reinterpret_cast<float2 *>(&(value))[0])
#define BFLOAT4_INPUT(value) (reinterpret_cast<const float2 *>(&(value))[0])

template <typename T>
__global__ void embedding_forward_kernel(const int64_t* input, const T* weight, T* output, int emb_size) {
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int offset = input[bx] * emb_size;
    if constexpr (std::is_same<T, float>::value) {
        FLOAT4_OUTPUT(output[bx * emb_size + 4 * tx]) = FLOAT4_INPUT(weight[offset + 4 * tx]);
    } else if constexpr (std::is_same<T, __hip_bfloat16>::value) {
        BFLOAT4_OUTPUT(output[bx * emb_size + 4 * tx]) = BFLOAT4_INPUT(weight[offset + 4 * tx]);
    }
}

void embedding_impl(torch::Tensor& output, const torch::Tensor& input, const torch::Tensor& weight) {
    int seq_len = input.numel();
    int emb_size = weight.size(1);
    dim3 block(emb_size / 4, 1, 1);
    dim3 grid(seq_len, 1, 1);
    c10::cuda::OptionalCUDAGuard device_guard(input.device());
    auto stream = at::cuda::getCurrentCUDAStream();

    if (weight.dtype() == torch::kBFloat16) {
        embedding_forward_kernel<__hip_bfloat16><<<grid, block, 0, stream>>>(
            reinterpret_cast<int64_t*>(input.data_ptr()), reinterpret_cast<__hip_bfloat16*>(weight.data_ptr()), reinterpret_cast<__hip_bfloat16*>(output.data_ptr()), emb_size);
    } else if (weight.dtype() == torch::kFloat32) {
        embedding_forward_kernel<float><<<grid, block, 0, stream>>>(
            reinterpret_cast<int64_t*>(input.data_ptr()), reinterpret_cast<float*>(weight.data_ptr()), reinterpret_cast<float*>(output.data_ptr()), emb_size);
    } else {
        throw std::runtime_error("Unsupported data type for embedding");
    }
}

// namespace infer {
// template <typename T>
// void EmbeddingOperator<T>::forward(const Tensor<int32_t>* input, const Tensor<T>* weight,
//                                    Tensor<T>* output) {
//     auto seq_len = input->shape()[0];
//     auto emb_size = weight->shape()[1];
//     // dim3 block(emb_size / 4, 1, 1);
//     // dim3 grid(seq_len, 1, 1);
//     // embedding_forward_kernel<T><<<grid, block, 0, input->getStream()>>>(
//     //     input->data_ptr(), weight->data_ptr(), output->data_ptr(), emb_size);
// }
// }